#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include "vector.h"
#include "config.h"

#include<stdio.h>

#undef DEBUG
//#define DEBUG 1

#define THREADS_PER_BLOCK 256
#define VECTOR_SIZE 3
#define BLOCK_WIDTH 16

__global__ void gpu_compute_accels(vector3* accels, vector3* hPos, double* mass) {

    // shared memory hopefully speeds things up
    __shared__ double shared_mass[BLOCK_WIDTH];
    __shared__ vector3 shared_hPos_x[BLOCK_WIDTH];
    __shared__ vector3 shared_hPos_y[BLOCK_WIDTH];

    // the entity being accelerated
    int entityIndex = blockIdx.x * BLOCK_WIDTH + threadIdx.x;

    // the entity causing the acceleration
    int otherEntityIndex = blockIdx.y * BLOCK_WIDTH + threadIdx.y;

    if (threadIdx.x == 0) {
        shared_mass[threadIdx.y] = mass[otherEntityIndex];
        shared_hPos_x[threadIdx.y][0] = hPos[otherEntityIndex][0];
        shared_hPos_x[threadIdx.y][1] = hPos[otherEntityIndex][1];
        shared_hPos_x[threadIdx.y][2] = hPos[otherEntityIndex][2];
    }
    if (threadIdx.y == 0) {
        shared_hPos_y[threadIdx.x][0] = hPos[entityIndex][0];
        shared_hPos_y[threadIdx.x][1] = hPos[entityIndex][1];
        shared_hPos_y[threadIdx.x][2] = hPos[entityIndex][2];
    }

    __syncthreads();

    if(entityIndex < NUMENTITIES && otherEntityIndex < NUMENTITIES) {

        if (entityIndex == otherEntityIndex) {
				FILL_VECTOR(accels[entityIndex * NUMENTITIES + otherEntityIndex],0,0,0);
			}
        else{
            vector3 distance;
            for (int k=0;k<3;k++) distance[k]=shared_hPos_y[threadIdx.x][k]-shared_hPos_x[threadIdx.y][k];
            double magnitude_sq=distance[0]*distance[0]+distance[1]*distance[1]+distance[2]*distance[2];
            double magnitude=sqrt(magnitude_sq);
            double accelmag=-1*GRAV_CONSTANT*shared_mass[threadIdx.y]/magnitude_sq;
            FILL_VECTOR(accels[entityIndex * NUMENTITIES + otherEntityIndex],accelmag*distance[0]/magnitude,accelmag*distance[1]/magnitude,accelmag*distance[2]/magnitude);
        }
    }
}

void compute_accels(vector3* h_values, vector3* d_values) {

    dim3 blockSize = dim3(BLOCK_WIDTH, BLOCK_WIDTH, 1);
    dim3 gridSize = dim3(ceil((float)NUMENTITIES / (float)BLOCK_WIDTH), ceil((float)NUMENTITIES / (float)BLOCK_WIDTH), 1);

    // call the kernel
    gpu_compute_accels<<<gridSize, blockSize>>>(d_values, d_hPos, d_mass);

    #ifdef DEBUG
    // copy the gpu acceleration matrix back to the host acceleration matrix
    hipMemcpy(h_values, d_values, sizeof(vector3) * NUMENTITIES * NUMENTITIES, hipMemcpyDeviceToHost);

    printf("GRID SIZE: %d\n", gridSize.x);

    // print out the matrix after GPU operation
    for (int i = 0; i < NUMENTITIES * NUMENTITIES; i++) {
        printf("%32.32f\n", h_values[i][0]);
        if(i % NUMENTITIES == 0)
            printf("\n");        
    }
    printf("\n");
    #endif
}

// from lecture slides could be useful
__global__ void sumNoncommSingleBlock(int *gArr, int *out, int arraySize) {
    int thIdx = threadIdx.x;
    __shared__ int shArr[THREADS_PER_BLOCK*2];
    __shared__ int offset;

    shArr[thIdx] = thIdx < arraySize ? gArr[thIdx] : 0;

    if(thIdx == 0)
        offset = THREADS_PER_BLOCK;
    __syncthreads();

    while(offset < arraySize) {
        shArr[thIdx + THREADS_PER_BLOCK] = thIdx + offset < arraySize ? gArr[thIdx] : 0;
        __syncthreads();
        if(thIdx == 0)
            offset += THREADS_PER_BLOCK;
        
        int sum = shArr[2*thIdx] + shArr[2*thIdx+1];
        __syncthreads();
        shArr[thIdx] = sum;
    }
    __syncthreads();

    for(int stride = 1; stride < THREADS_PER_BLOCK; stride *= 2) {
        int arrIdx = thIdx*stride*2;
        if(arrIdx + stride < THREADS_PER_BLOCK)
            shArr[arrIdx] += shArr[arrIdx + stride];
        __syncthreads();
    }
    if(thIdx == 0)
        *out = shArr[0];
}

__global__ void sumOneVectorPerBlock(int *gArr, int *out, int arraySize) {
    int thIdx = threadIdx.x;
    int bIdx = blockIdx.x;

    __shared__ int shArr[THREADS_PER_BLOCK * 2];
    __shared__ int offset;

    shArr[thIdx] = thIdx < arraySize ? gArr[bIdx * arraySize + thIdx] : 0;

    if (thIdx == 0)
        offset = blockDim.x;
    __syncthreads();

    while (offset < arraySize) {

        shArr[thIdx + THREADS_PER_BLOCK] =
            thIdx + offset < arraySize ? gArr[bIdx * arraySize + thIdx + offset] : 0;
        __syncthreads();

        if (thIdx == 0)
            offset += THREADS_PER_BLOCK;

        int sum = shArr[2 * thIdx] + shArr[2 * thIdx + 1];
        __syncthreads();
        shArr[thIdx] = sum;
    }
    __syncthreads();

    for (int stride = 1; stride < THREADS_PER_BLOCK; stride *= 2) {
        int arrIdx = thIdx * stride * 2;
        if (arrIdx + stride < THREADS_PER_BLOCK)
            shArr[arrIdx] += shArr[arrIdx + stride];
        __syncthreads();
    }

    if (thIdx == 0)
        out[bIdx] = shArr[0];
}

void sumAccelerations(int *d_input, int *d_output, int arraySize, int numVectors) {

    dim3 gridSize = dim3(numVectors, 1, 1);
    dim3 blockSize = dim3(THREADS_PER_BLOCK, 1, 1);

    sumOneVectorPerBlock<<<gridSize, blockSize>>>(d_input, d_output, arraySize);

    hipDeviceSynchronize();
    hipError_t hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(hipError_t));
    }
}

__global__ void gpu_advance_simulation(vector3* d_values, vector3* hVel, vector3* hPos) {
    
    int entityIndex = (blockIdx.x * blockDim.x + threadIdx.x) * NUMENTITIES;

    for(int i = entityIndex + 1; i < entityIndex + NUMENTITIES; i++) {
        d_values[entityIndex][0] += d_values[i][0];
        d_values[entityIndex][1] += d_values[i][1];
        d_values[entityIndex][2] += d_values[i][2];
    }

    for (int k = 0; k < 3; k++){
        hVel[blockIdx.x * blockDim.x + threadIdx.x][k]+=d_values[entityIndex][k]*INTERVAL;
        hPos[blockIdx.x * blockDim.x + threadIdx.x][k]+=hVel[blockIdx.x * blockDim.x + threadIdx.x][k]*INTERVAL;
    }
}

void advance_simulation(vector3* h_values, vector3* d_values, vector3* d_hVel, vector3* d_hPos) {

    dim3 blockSize = dim3(256, 1, 1);
    dim3 gridSize = dim3(ceil((float)NUMENTITIES / 256), 1, 1);

    gpu_advance_simulation<<<gridSize, blockSize>>>(d_values, d_hVel, d_hPos);

    hipMemcpy(hPos, d_hPos, sizeof(vector3) * NUMENTITIES, hipMemcpyDeviceToHost);
    hipMemcpy(hVel, d_hVel, sizeof(vector3) * NUMENTITIES, hipMemcpyDeviceToHost);

    #ifdef DEBUG
    printf("GRID SIZE: %d\n", gridSize.x);
    // copy the gpu acceleration matrix back to the host acceleration matrix
    hipMemcpy(h_values, d_values, sizeof(vector3) * NUMENTITIES * NUMENTITIES, hipMemcpyDeviceToHost);

    // print out the matrix after GPU operation
    for (int i = 0; i < NUMENTITIES; i+=1) {
        printf("%10.32f\n",hVel[i][0]);
        if (i % NUMENTITIES == 0) {
            //printf("%32.32f\n", h_values[i][0]);
            //printf("%32.1f %32.1f %32.32f\n", h_values[i][0], h_values[i][1], h_values[i][2]);
        }
    }
    printf("\n");
    #endif
}

//compute: Updates the positions and locations of the objects in the system based on gravity.
//Parameters: None
//Returns: None
//Side Effect: Modifies the hPos and hVel arrays with the new positions and accelerations after 1 INTERVAL
void compute(){

	// make an acceleration matrix which is NUMENTITIES squared in size
	vector3* h_values=(vector3*)malloc(sizeof(vector3)*NUMENTITIES*NUMENTITIES);
	vector3** h_accels=(vector3**)malloc(sizeof(vector3*)*NUMENTITIES);
	for (int i = 0; i < NUMENTITIES; i++)
		h_accels[i]=&h_values[i*NUMENTITIES];

    for (int i = 0; i < NUMENTITIES; i++) {
        for(int j = 0; j < NUMENTITIES; j++) {
            for(int k = 0; k < VECTOR_SIZE; k++) {
                h_accels[i][j][k] = 2.0;
            }
        }
    }

    // create that accleration "matrix" on the GPU
    vector3* d_values;
    hipMalloc((void **)&d_values, sizeof(vector3) * NUMENTITIES * NUMENTITIES);

    // create the positions array on the GPU
    hipMalloc((void **)&d_hPos, sizeof(vector3) * NUMENTITIES);
    hipMemcpy(d_hPos, hPos, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);

    // create the mass array on the GPU
    hipMalloc((void **)&d_mass, sizeof(double) * NUMENTITIES);
    hipMemcpy(d_mass, mass, sizeof(double) * NUMENTITIES, hipMemcpyHostToDevice);

    // create the velocity array on the GPU
	hipMalloc((void **)&d_hVel, sizeof(vector3) * NUMENTITIES);
    hipMemcpy(d_hVel, hVel, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);

    //compute_accels(h_values, d_values);
    //advance_simulation(h_values, d_values, d_hVel, d_hPos);

    // START TEST ZONE
    int size = 6;
    
    int *h_input = (int*)malloc(sizeof(int) * size * size);
    for(int i = 0; i < size * size; i++) {
        h_input[i] = i;
    }

    int *h_output = (int*)malloc(sizeof(int) * size);
    for(int i = 0; i < size; i++) {
        h_output[i] = 0;
    }
/*     int *h_output = (int*)malloc(sizeof(int)); */

    int * d_input;
    hipMalloc((void **)&d_input, sizeof(int) * size * size);
    hipMemcpy(d_input, h_input, sizeof(int) * size * size, hipMemcpyHostToDevice);

    int * d_output;
    hipMalloc((void **)&d_output, sizeof(int) * size);
/*     hipMalloc((void **)&d_output, sizeof(int)); */

    sumAccelerations(d_input, d_output, size, size*size);
/*     sumNoncommSingleBlock<<<1, THREADS_PER_BLOCK>>>(d_input, d_output, size*size); */

    hipMemcpy(h_output, d_output, sizeof(int) * size, hipMemcpyDeviceToHost);
    for (int i = 0; i < size; i++) {
        printf("%d\n", h_output[i]);
    }

/*     hipMemcpy(h_output, d_output, sizeof(int), hipMemcpyDeviceToHost);
    printf("%d\n", *h_output); */

    // END TEST ZONE

    free(h_accels);
	free(h_values);

    hipFree(d_values);
    hipFree(d_hVel);
    hipFree(d_hPos);
    hipFree(d_mass);
}
