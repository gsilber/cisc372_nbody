#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include "vector.h"
#include "config.h"

#include<stdio.h>

/* __global__ void compute_accels(vector3* accels) {

    for(int i = 0; i < NUMENTITIES * NUMENTITIES; i++) {
        accels[i][0] += 1.0;
    }
} */

__global__ void compute_accels(vector3** accels) {

    for(int i = 0; i < NUMENTITIES; i++) {
        for(int j = 0; j < NUMENTITIES; j++) {
            accels[i][j][0] += 3.0;
        }
    }
}

//compute: Updates the positions and locations of the objects in the system based on gravity.
//Parameters: None
//Returns: None
//Side Effect: Modifies the hPos and hVel arrays with the new positions and accelerations after 1 INTERVAL
void compute(){

	// make an acceleration matrix which is NUMENTITIES squared in size
	vector3* h_values=(vector3*)malloc(sizeof(vector3)*NUMENTITIES*NUMENTITIES);
	vector3** h_accels=(vector3**)malloc(sizeof(vector3*)*NUMENTITIES);
	for (int i = 0; i < NUMENTITIES; i++)
		h_accels[i]=&h_values[i*NUMENTITIES];

    for (int i = 0; i < NUMENTITIES; i++) {
        for(int j = 0; j < NUMENTITIES; j++) {
            h_accels[i][j][0] = 2.0;
        }
    }

    // create that accleration matrix on the GPU
    vector3* d_values;
    hipMalloc((void **)&d_values, sizeof(vector3) * NUMENTITIES * NUMENTITIES);

    vector3** d_accels;
    hipMalloc((void **)&d_accels, sizeof(vector3*) * NUMENTITIES);

    for (int i = 0; i < NUMENTITIES; i++) {
        vector3* d_accel_row;
        hipMalloc((void **)&d_accel_row, sizeof(vector3) * NUMENTITIES);
        hipMemcpy(d_accels+i, &d_accel_row, sizeof(vector3*), hipMemcpyHostToDevice);
        hipMemcpy(d_accel_row, h_accels[i], sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);
    }

    //hipMemcpy(d_values, h_values, sizeof(vector3) * NUMENTITIES * NUMENTITIES, hipMemcpyHostToDevice);

    //compute_accels<<<1,1>>>(d_values);
    compute_accels<<<1,1>>>(d_accels);

    printf("got through kernel\n");

    for (int i = 0; i < NUMENTITIES; i++) {
        vector3* d_accel_row;

        hipMemcpy(&d_accel_row, d_accels+i, sizeof(vector3*), hipMemcpyDeviceToHost);
        printf("copied the row address back\n");
        hipMemcpy(h_accels[i], d_accel_row, sizeof(vector3) * NUMENTITIES, hipMemcpyDeviceToHost);
        printf("copied the row data back\n");
    }

//    hipMemcpy(h_values, d_values, sizeof(vector3) * NUMENTITIES * NUMENTITIES, hipMemcpyDeviceToHost);
    
/*     for(int i = 0; i < NUMENTITIES * NUMENTITIES; i++) {
        if(i % NUMENTITIES == 0) printf("\n");
        printf("%f ", h_values[i][0]);
    } */


    for (int i = 0; i < NUMENTITIES; i++) {
        for(int j = 0; j < NUMENTITIES; j++) {
            printf("%1.1f ", h_accels[i][j][0]);
        }
        printf("\n");
    }
    printf("\n");

    // allocate device memory of velocity, position, and mass
	hipMalloc((void **)&d_hVel, sizeof(vector3) * NUMENTITIES);
    hipMemcpy(d_hVel, hVel, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_hPos, sizeof(vector3) * NUMENTITIES);
    hipMemcpy(d_hPos, hPos, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_mass, sizeof(double) * NUMENTITIES);
    hipMemcpy(d_mass, mass, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);

    //compute_accels<<<1,1>>>(d_hPos);
    //hipMemcpy(hPos, d_hPos, sizeof(vector3) * NUMENTITIES, hipMemcpyDeviceToHost);
	
    //first compute the pairwise accelerations.  Effect is on the first argument.
	for (int i = 0; i < NUMENTITIES; i++){
		for (int j = 0; j < NUMENTITIES; j++){
			if (i==j) {
				FILL_VECTOR(h_accels[i][j],0,0,0);
			}
			else{
				vector3 distance;
				for (int k = 0; k < 3; k++) distance[k]=hPos[i][k]-hPos[j][k];
				double magnitude_sq=distance[0]*distance[0]+distance[1]*distance[1]+distance[2]*distance[2];
				double magnitude=sqrt(magnitude_sq);
				double accelmag=-1*GRAV_CONSTANT*mass[j]/magnitude_sq;
				FILL_VECTOR(h_accels[i][j],accelmag*distance[0]/magnitude,accelmag*distance[1]/magnitude,accelmag*distance[2]/magnitude);
			}
		}
	}

    //sum up the rows of our matrix to get effect on each entity, then update velocity and position.
	for (int i = 0; i < NUMENTITIES; i++){
		vector3 accel_sum={0,0,0};
		for (int j = 0; j < NUMENTITIES; j++){
			for (int k = 0; k < 3; k++)
				accel_sum[k]+=h_accels[i][j][k];
		}
		//compute the new velocity based on the acceleration and time interval
		//compute the new position based on the velocity and time interval
		for (int k = 0; k < 3; k++){
			hVel[i][k]+=accel_sum[k]*INTERVAL;
			hPos[i][k]+=hVel[i][k]*INTERVAL;
		}
	}
	
    free(h_accels);
	free(h_values);

    hipFree(d_hVel);
    hipFree(d_hPos);
    hipFree(d_mass);
}
