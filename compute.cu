#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include "vector.h"
#include "config.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

//compute: Updates the positions and locations of the objects in the system based on gravity.
//Parameters: None
//Returns: None
//Side Effect: Modifies the hPos and hVel arrays with the new positions and accelerations after 1 INTERVAL
//__global__ void compute(vector3 *hVel,vector3 *hPos,double *mass)
__global__ void compute(vector3 *d_hVel,vector3 *d_hPos,double *d_mass,vector3* values,vector3** accels){
	//make an acceleration matrix which is NUMENTITIES squared in size;
	int j,k;
	int i =  blockDim.x * blockIdx.x +  threadIdx.x;
	//int i = blockDim.x * blockIdx.x +blockIdx.x + (16*threadIdx.y);
	//vector3* values=(vector3*)malloc(sizeof(vector3)*NUMENTITIES*NUMENTITIES);
	//vector3** accels=(vector3**)malloc(sizeof(vector3*)*NUMENTITIES);
	//__shared__ vector3* values[NUMENTITIES*NUMENTITIES];
	//__shared__ vector3** accels[NUMENTITIES];
	//printf("bx:%d   by: %d  tx: %d ty: %d\n",blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y);
	//printf("test:%d\n",i);
	if (i < NUMENTITIES){
		accels[i]=&values[i*NUMENTITIES];
	}
	//first compute the pairwise accelerations.  Effect is on the first argument.
	if (i < NUMENTITIES){
		for (j=0;j<NUMENTITIES;j++){
			//printf("test: %d \n",j);
			if (i==j) {
				FILL_VECTOR(accels[i][j],0,0,0);
			}
			else{
				vector3 distance;
				for (k=0;k<3;k++) distance[k]=d_hPos[i][k]-d_hPos[j][k];
				double magnitude_sq=distance[0]*distance[0]+distance[1]*distance[1]+distance[2]*distance[2];
				double magnitude=sqrt(magnitude_sq);
				double accelmag=-1*GRAV_CONSTANT* d_mass[j]/magnitude_sq;
				FILL_VECTOR(accels[i][j],accelmag*distance[0]/magnitude,accelmag*distance[1]/magnitude,accelmag*distance[2]/magnitude);

			}

		}
	}
	//sum up the rows of our matrix to get effect on each entity, then update velocity and position.
	if (i < NUMENTITIES){
		vector3 accel_sum={0,0,0};
		for (j=0;j<NUMENTITIES;j++){
			for (k=0;k<3;k++)
				accel_sum[k]+=accels[i][j][k];
		}
		//compute the new velocity based on the acceleration and time interval
		//compute the new position based on the velocity and time interval
		for (k=0;k<3;k++){
			d_hVel[i][k]+=accel_sum[k]*INTERVAL;
			d_hPos[i][k]=d_hVel[i][k]*INTERVAL;
		}
	}

	//free(accels);
	//free(values);
}
