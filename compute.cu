#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include "vector.h"
#include "config.h"

#include <stdio.h>

//#define DEBUGACCEL 1
//#define DEBUGSUM 1


#define BLOCK_WIDTH_ACCELS 16

#define SUM_TOTAL_THREADS 1

// DO NOT CHANGE THE VECTOR SIZE
#define VECTORSIZE 3

__global__ void compute_accels(vector3 *accels, vector3* pos, double* mass) {

    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y;

    __shared__ double distances[BLOCK_WIDTH_ACCELS][BLOCK_WIDTH_ACCELS][VECTORSIZE];

    if(i < NUMENTITIES && j < NUMENTITIES) {
        distances[threadIdx.x][threadIdx.y][threadIdx.z] = pos[i][threadIdx.z] - pos[j][threadIdx.z];
    }
    __syncthreads();

    if(i < NUMENTITIES && j < NUMENTITIES) {

        if (i == j) {
            accels[i * NUMENTITIES + j][threadIdx.z] = 0.0;
        }
        else{
            double magnitude_sq = ( 
                distances[threadIdx.x][threadIdx.y][0] * distances[threadIdx.x][threadIdx.y][0] + 
                distances[threadIdx.x][threadIdx.y][1] * distances[threadIdx.x][threadIdx.y][1] + 
                distances[threadIdx.x][threadIdx.y][2] * distances[threadIdx.x][threadIdx.y][2]
            );

            double magnitude = sqrt(magnitude_sq);
            double accelmag =- 1 * GRAV_CONSTANT * mass[j] / magnitude_sq;
            accels[i * NUMENTITIES + j][threadIdx.z] = accelmag * distances[threadIdx.x][threadIdx.y][threadIdx.z] / magnitude;
        }
    }
}

__global__ void sumOneVectorComponentPerBlock(vector3* gArr, vector3* out) {
    __shared__ double shArr[SUM_TOTAL_THREADS * 2];
    __shared__ int offset;

    shArr[threadIdx.x] = threadIdx.x < NUMENTITIES ? gArr[blockIdx.x * NUMENTITIES + threadIdx.x][blockIdx.y] : 0;

    if (threadIdx.x == 0)
        offset = blockDim.x;
    __syncthreads();

    while (offset < NUMENTITIES) {

        shArr[threadIdx.x + SUM_TOTAL_THREADS] = threadIdx.x + offset < NUMENTITIES ? gArr[blockIdx.x * NUMENTITIES + threadIdx.x + offset][blockIdx.y] : 0;
        __syncthreads();

        if (threadIdx.x == 0)
            offset += SUM_TOTAL_THREADS;
 
        double sum = shArr[2 * threadIdx.x] + shArr[2 * threadIdx.x + 1];
        __syncthreads();

        shArr[threadIdx.x] = sum;
    }
    __syncthreads();

    for (int stride = 1; stride < SUM_TOTAL_THREADS; stride *= 2) {
        __syncthreads();
        int arrIdx = threadIdx.x * stride * 2;
        if (arrIdx + stride < SUM_TOTAL_THREADS) {
            shArr[arrIdx] += shArr[arrIdx + stride];
        }
    }

    __syncthreads();
    if (threadIdx.x == 0) {
        out[blockIdx.x][blockIdx.y] = shArr[0];
    }
}


__global__ void advance_time(vector3* accel, vector3* vel, vector3* pos) {
    vel[blockIdx.x][threadIdx.x] += accel[blockIdx.x][threadIdx.x]*INTERVAL;
    pos[blockIdx.x][threadIdx.x] += vel[blockIdx.x][threadIdx.x]*INTERVAL;
}

//compute: Updates the positions and locations of the objects in the system based on gravity.
//Parameters: None
//Returns: None
//Side Effect: Modifies the hPos and hVel arrays with the new positions and accelerations after 1 INTERVAL
void compute(){

    dim3 blockSize = dim3(16, 16, 3);
    dim3 gridSize = dim3(ceil((double)NUMENTITIES / (double)blockSize.x), ceil((double)NUMENTITIES / (double)blockSize.y), 1);

    compute_accels<<<gridSize, blockSize>>>(d_hAccels, d_hPos, d_hmass);

    #ifdef DEBUGACCEL
    printf("printing accels\n");
    hipMemcpy(hAccels, d_hAccels, sizeof(vector3) * NUMENTITIES * NUMENTITIES, hipMemcpyDeviceToHost);
    for(int i = 0; i < NUMENTITIES * NUMENTITIES; i ++) {
        
        if(i % NUMENTITIES == 0) {
            printf("\n");
        }
        printf("%.32f\n", hAccels[i][0]);
    }
    printf("\n"); 
    #endif

    blockSize = dim3(SUM_TOTAL_THREADS, 1, 1);
    gridSize = dim3(NUMENTITIES, VECTORSIZE, 1);

    hipDeviceSynchronize();
    //sumOneVectorPerBlock<<<gridSize, blockSize>>>(d_hAccels, d_output, NUMENTITIES);

    sumOneVectorComponentPerBlock<<<gridSize, blockSize>>>(d_hAccels, d_output);
    hipDeviceSynchronize();

    #ifdef DEBUGSUM
    printf("printing sums\n");
    vector3 *h_output = (vector3*)malloc(sizeof(vector3) * NUMENTITIES);

    hipMemcpy(h_output, d_output, sizeof(vector3) * NUMENTITIES, hipMemcpyDeviceToHost);
    for(int i = 0; i < NUMENTITIES; i++) {
        printf("%.32f %.32f %.32f\n", h_output[i][0], h_output[i][1], h_output[i][2]);
    }

    hipError_t hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(hipError_t));
    }
    #endif

    blockSize = dim3(VECTORSIZE, 1, 1);
    gridSize = dim3(NUMENTITIES, 1, 1);

    hipDeviceSynchronize();
    advance_time<<<gridSize, blockSize>>> (d_output, d_hVel, d_hPos);
}