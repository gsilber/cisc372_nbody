#include "hip/hip_runtime.h"
/*
	CISC372 Assignment 4: Lost in space 
	Contributors: Patrick Harris, Robert Reardon
	File: nbody.cu
*/
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "vector.h"
#include "config.h"
#include "planets.h"
#include "compute.h"

// represents the objects in the system.  Global variables
vector3 *hVel, *d_hvel;
vector3 *hPos, *d_hpos;
double *mass, *d_mass; //dmass: to be passed onto the device memory.


// Initialize accels array using CUDA kernel function.
__global__ void initializeAccels(vector3 **accels, vector3 *values, int numEntities) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < numEntities) {
        accels[idx] = &values[idx*numEntities];
    }
}


// CUDA kernel to compute pairwise accelerations
__global__ void computeAccels(vector3** accels, vector3* hPos, double* mass) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j;
    vector3 distance;
    double magnitude_sq, magnitude, accelmag;
    if (i < NUMENTITIES) {
        for (j = 0; j < NUMENTITIES; j++) {
            if (i == j) {
                FILL_VECTOR(accels[i][j], 0, 0, 0);
            } else {
                for (int k = 0; k < 3; k++) {
                    distance[k] = hPos[i][k] - hPos[j][k];
                }
                magnitude_sq = distance[0] * distance[0] + distance[1] * distance[1] + distance[2] * distance[2];
                magnitude = sqrt(magnitude_sq);
                accelmag = -1 * GRAV_CONSTANT * mass[j] / magnitude_sq;
                FILL_VECTOR(accels[i][j], accelmag * distance[0] / magnitude, accelmag * distance[1] / magnitude, accelmag * distance[2] / magnitude);
            }
        }
    }
}

// CUDA kernel to sum up the rows of the acceleration matrix
__global__ void sumAccels(vector3** accels, vector3* accel_sum) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < NUMENTITIES) {
        for (int j = 0; j < NUMENTITIES; j++) {
            for (int k = 0; k < 3; k++) {
                accel_sum[i][k] += accels[i][j][k];
            }
        }
    }
}

// CUDA kernel to update velocity and position
__global__ void updateVelPos(vector3* hVel, vector3* hPos, vector3* accel_sum) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < NUMENTITIES) {
        for (int k = 0; k < 3; k++) {
            hVel[i][k] += accel_sum[i][k] * INTERVAL;
            hPos[i][k] += hVel[i][k] * INTERVAL;
        }
    }
}

//initHostMemory: Create storage for numObjects entities in our system
//Parameters: numObjects: number of objects to allocate
//Returns: None
//Side Effects: Allocates memory in the hVel, hPos, and mass global variables
void initHostMemory(int numObjects)
{
	hVel = (vector3 *)malloc(sizeof(vector3) * numObjects);
	hPos = (vector3 *)malloc(sizeof(vector3) * numObjects);
	mass = (double *)malloc(sizeof(double) * numObjects);
}

//freeHostMemory: Free storage allocated by a previous call to initHostMemory
//Parameters: None
//Returns: None
//Side Effects: Frees the memory allocated to global variables hVel, hPos, and mass.
void freeHostMemory()
{
	free(hVel);
	free(hPos);
	free(mass);
}

//planetFill: Fill the first NUMPLANETS+1 entries of the entity arrays with an estimation
//				of our solar system (Sun+NUMPLANETS)
//Parameters: None
//Returns: None
//Fills the first 8 entries of our system with an estimation of the sun plus our 8 planets.
void planetFill(){
	int i,j;
	double data[][7]={SUN,MERCURY,VENUS,EARTH,MARS,JUPITER,SATURN,URANUS,NEPTUNE};
	for (i=0;i<=NUMPLANETS;i++){
		for (j=0;j<3;j++){
			hPos[i][j]=data[i][j];
			hVel[i][j]=data[i][j+3];
		}
		mass[i]=data[i][6];
	}
}

//randomFill: FIll the rest of the objects in the system randomly starting at some entry in the list
//Parameters: 	start: The index of the first open entry in our system (after planetFill).
//				count: The number of random objects to put into our system
//Returns: None
//Side Effects: Fills count entries in our system starting at index start (0 based)
void randomFill(int start, int count)
{
	int i, j;// c = start;
	for (i = start; i < start + count; i++)
	{
		for (j = 0; j < 3; j++)
		{
			hVel[i][j] = (double)rand() / RAND_MAX * MAX_DISTANCE * 2 - MAX_DISTANCE;
			hPos[i][j] = (double)rand() / RAND_MAX * MAX_VELOCITY * 2 - MAX_VELOCITY;
			mass[i] = (double)rand() / RAND_MAX * MAX_MASS;
		}
	}
}

//printSystem: Prints out the entire system to the supplied file
//Parameters: 	handle: A handle to an open file with write access to prnt the data to
//Returns: 		none
//Side Effects: Modifies the file handle by writing to it.
void printSystem(FILE* handle){
	int i,j;
	for (i=0;i<NUMENTITIES;i++){
		fprintf(handle,"pos=(");
		for (j=0;j<3;j++){
			fprintf(handle,"%lf,",hPos[i][j]);
		}
		printf("),v=(");
		for (j=0;j<3;j++){
			fprintf(handle,"%lf,",hVel[i][j]);
		}
		fprintf(handle,"),m=%lf\n",mass[i]);
	}
}

int main(int argc, char **argv)
{
	clock_t t0=clock();
	int t_now;
	//srand(time(NULL));
	srand(1234);
	initHostMemory(NUMENTITIES);
	planetFill();
	randomFill(NUMPLANETS + 1, NUMASTEROIDS);
	//now we have a system.

	#ifdef DEBUG
	printSystem(stdout);
	#endif

	// Allocating memory on device
	hipMalloc((void**)&d_hvel, sizeof(vector3)*NUMENTITIES);
	hipMalloc((void**)&d_hpos, sizeof(vector3)*NUMENTITIES);
	hipMalloc((void**)&d_mass, sizeof(double)*NUMENTITIES);

	// Copying data from host to device
	hipMemcpy(d_hvel, hVel, sizeof(vector3)*NUMENTITIES, hipMemcpyHostToDevice);
	hipMemcpy(d_hpos, hPos, sizeof(vector3)*NUMENTITIES, hipMemcpyHostToDevice);
	hipMemcpy(d_mass, mass, sizeof(double)*NUMENTITIES, hipMemcpyHostToDevice);

	int threads_per_block = 256;
    	int num_blocks = (NUMENTITIES + threads_per_block - 1) / threads_per_block;
	
	vector3 *d_values;
	hipMalloc((void **)&d_values, sizeof(vector3)*NUMENTITIES*NUMENTITIES);
	vector3 **d_accels;
	hipMalloc((void ***)&d_accels, sizeof(vector3*)*NUMENTITIES);
	initializeAccels<<<num_blocks, threads_per_block>>>(d_accels, d_values, NUMENTITIES);
	hipDeviceSynchronize();
	
	vector3 h_accel_sum = {0, 0, 0};
	vector3* d_accel_sum;
	hipMalloc((void **)&d_accel_sum, sizeof(vector3));

	
	// Call Kernal function for each INTERVAL
	for(t_now = 0; t_now < DURATION; t_now+= INTERVAL){
		computeAccels<<<num_blocks, threads_per_block>>>(d_accels, d_hpos, d_mass);

		hipMemcpy(d_accel_sum, &h_accel_sum, sizeof(vector3), hipMemcpyHostToDevice);

		sumAccels<<<num_blocks, threads_per_block>>>(d_accels, d_accel_sum);
		updateVelPos<<<num_blocks, threads_per_block>>>(d_hvel, d_hpos, d_accel_sum);		
	}

	// Copying data from device to host
	hipMemcpy(hVel, d_hvel, sizeof(vector3)*NUMENTITIES, hipMemcpyDeviceToHost);
	hipMemcpy(hPos, d_hpos, sizeof(vector3)*NUMENTITIES, hipMemcpyDeviceToHost);
	
	// Deallocating memory on device
	hipFree(d_accel_sum);
	hipFree(d_hvel);
	hipFree(d_hpos);
	hipFree(d_mass);
	hipFree(d_accels);
	hipFree(d_values);

	clock_t t1=clock()-t0;

	#ifdef DEBUG
	printSystem(stdout);
	#endif

	printf("This took a total time of %f seconds\n",(double)t1/CLOCKS_PER_SEC);

	freeHostMemory();
}