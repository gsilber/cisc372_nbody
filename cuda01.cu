#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "vector.h"

__global__ void sum(vector3 *x,int N){
    int tid =  blockDim.x * blockIdx.x + threadIdx.x;
    if(tid < N){
        x[1][tid] = tid;
    }
}
int main(){
    int N = 130;
    int nbytes = sizeof(vector3) * N;
    vector3 *hx, *dx;
    int i;


    hipMalloc((void**)&dx, nbytes);
    hx = (vector3 *)malloc(nbytes);
    for(i = 0; i< N; i++){
        hx[1][i] = 1;
        printf("%g\n",hx[1][i]);
    }
    
    hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);
    dim3 dimGrid(3);
    dim3 dimBlock(64);
    sum<<<dimGrid,dimBlock>>>(dx,N);
    //hipDeviceSynchronize();
    hipMemcpy(hx, dx, nbytes, hipMemcpyDeviceToHost);
    for(i = 0; i< N; i++){
        printf("%g\n",hx[1][i]);
    }
    hipFree(dx);
    free(hx);
    return 0;
}
