#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "vector.h"

__global__ void sum(vector3 *dx,int N){
    int tid =  blockDim.x * blockIdx.x + threadIdx.x;
    if(tid < N){
        dx[1][tid] = tid;
        //printf("%g\n",x[1][tid]);
    }
}
int main(){
    int N = 64;
    int nbytes = sizeof(vector3) * N;
    vector3 *hx, *dx;
    int i;

    hipMallocManaged((void**)&dx, nbytes);
    //hipMalloc((void**)&dx, nbytes);
    hx = (vector3 *)malloc(nbytes);
    for(i = 0; i< N; i++){
        dx[1][i] = 1;
        printf("%g\n",dx[1][i]);
    }
    //hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);
    dim3 dimGrid(1);
    dim3 dimBlock(64);
    sum<<<dimGrid,dimBlock>>>(dx,N);
    printf("CCCCCCCCCCCCCCCCCCC\n");
    hipDeviceSynchronize();
    //hipMemcpy(hx, dx, nbytes, hipMemcpyDeviceToHost);
    for(i = 0; i< N; i++){
        printf("%g\n",dx[1][i]);
    }
    hipFree(dx);
    free(hx);
    return 0;
}
